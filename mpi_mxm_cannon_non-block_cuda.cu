#include "hip/hip_runtime.h"
// Author: Zhi Shang <zhishang72@gmail.com>

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <math.h>
#include <mpi.h>

#define N 1024*4

#define BLOCK_SIZE 32
//function without shared memory at GPU
__global__ 
void cuda_mxm_noshare(double *a, double *b, double *c, int n)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if( col < n && row < n) 
    {
        c[row * n + col] = 0;
        for(int i = 0; i < n; i++) 
        {
            c[row * n + col] += a[row * n + i] * b[i * n + col];
        }
    }
}
//function with shared memory at GPU
__global__ 
void cuda_mxm_share(double *a, double *b, double *c, int n)
{
    __shared__ double sub_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double sub_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int idx_a, idx_b, c_val = 0;

    for (int x = 0; x < gridDim.x; x++)
    {
        idx_a = row * n + x * BLOCK_SIZE + threadIdx.x;
        sub_a[threadIdx.y][threadIdx.x] = 0.0;
        if(idx_a < n*n)
        {
            // n may not be fully divided by BLOCK_SIZE
            sub_a[threadIdx.y][threadIdx.x] = a[idx_a];
        }
        idx_b = (x * BLOCK_SIZE + threadIdx.y) * n + col;
        sub_b[threadIdx.y][threadIdx.x] = 0.0;
        if(idx_a < n*n && idx_b < n*n)
        {
            // n may not be fully divided by BLOCK_SIZE
            sub_b[threadIdx.y][threadIdx.x] = b[idx_b];
        }
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; k++)
        {
            c_val += sub_a[threadIdx.y][k] * sub_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    if(row < n && col < n)
    {
        c[row * n + col] = c_val;
    }
}

int main(int argc, char *argv[])
{
    MPI_Comm cannon_comm;
    MPI_Status status;
    MPI_Request req1,req2,req3,req4;
    int rank,size;
    int shift,i,j;
    int dims[2], periods[2];
    int left,right,up,down;
    double *buf=NULL,*tmp=NULL;
    double start,tloc,tmax;
    unsigned int iseed=0;

    MPI_Init(&argc,&argv);
    MPI_Comm_rank(MPI_COMM_WORLD,&rank);
    MPI_Comm_size(MPI_COMM_WORLD,&size);

    srand(iseed);

    dims[0]=0;    dims[1]=0;
    periods[0]=1; periods[1]=1;
    MPI_Dims_create(size,2,dims);
    if (dims[0]!=dims[1])
    {
        if(rank==0) printf("The number of processors must be a square.\n");
        MPI_Finalize();
        return 0;
    }

    int Nl=N/dims[0];
    buf=(double*)malloc(Nl*Nl*sizeof(double));
    
    double *A=NULL,*B=NULL,*C=NULL;    
    hipMallocManaged((void **) &A, sizeof(double)*Nl*Nl);
    hipMallocManaged((void **) &B, sizeof(double)*Nl*Nl);
    hipMallocManaged((void **) &C, sizeof(double)*Nl*Nl);

    unsigned int grid_rows = (Nl + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (Nl + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    
    if(rank==0) printf("MPI processors: %4i\n",size);

    MPI_Cart_create(MPI_COMM_WORLD,2,dims,periods,1,&cannon_comm);
    MPI_Cart_shift(cannon_comm,0,1,&left,&right);
    MPI_Cart_shift(cannon_comm,1,1,&up,&down);

    start=MPI_Wtime();
    for (i=0; i<Nl; i++)
    {
        for (j=0; j<Nl; j++)
        {
            A[i*Nl+j]=5-(int)( 10.0 * rand() / ( RAND_MAX + 1.0 ) );
            B[i*Nl+j]=5-(int)( 10.0 * rand() / ( RAND_MAX + 1.0 ) );
        }
    }
       
    for (shift=0; shift<dims[0]; shift++)
    {
    // Matrix multiplication
        //cuda_mxm_share<<<dimGrid, dimBlock>>>(A, B, C, Nl);     
        cuda_mxm_noshare<<<dimGrid, dimBlock>>>(A, B, C, Nl);
        // Transefr results from device to host 
        hipDeviceSynchronize();
   // Communication left - right
        MPI_Irecv(buf, Nl*Nl, MPI_DOUBLE, left, 100, cannon_comm, &req1);
        MPI_Isend(A, Nl*Nl, MPI_DOUBLE, right, 100, cannon_comm, &req2);
        tmp=buf; buf=A; A=tmp; 
        
   // Communication up - down
        MPI_Irecv(buf, Nl*Nl, MPI_DOUBLE, up, 200, cannon_comm, &req3);
        MPI_Isend(B, Nl*Nl, MPI_DOUBLE, down, 200, cannon_comm, &req4);
        tmp=buf; buf=B; B=tmp;
    }
    MPI_Wait(&req1, &status);
    MPI_Wait(&req2, &status);   
    MPI_Wait(&req3, &status);
    MPI_Wait(&req4, &status);
    tloc = MPI_Wtime() - start;
        
    MPI_Reduce(&tloc, &tmax, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);
    if(rank==0) printf("CPU time: %.4fs\n",tmax);
    hipFree(A); hipFree(B); hipFree(C); free(buf);
    MPI_Finalize();
    return 0;
}


